#include "hip/hip_runtime.h"

#include "utils.h"
#include <stdio.h>


__global__ void gaussian_blur(const unsigned char* const inputChannel,
                    unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth,const int s, int oRows, int oCols)
{
  // TODO
  
  // NOTE: Be sure to compute any intermediate results in floating point
  // before storing the final result as unsigned char.

  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  //
 
    
int x=blockIdx.x * blockDim.x + threadIdx.x;
    int y=blockIdx.y * blockDim.y + threadIdx.y;
    const int thread_1D_pos = y * oCols + x;

   if ( x>=oCols ||y>=oRows )
   {
       return;
   }
   float sum=0.0f;
//printf("gloc=%d =>threadId.x=%d,threadId.y=%d,blockId.x=%d,blocId.y=%d,position.x=%d,position.y=%d\n",thread_1D_pos,threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,threadIdx.x +(blockIdx.x* blockDim.x),threadIdx.y +(blockIdx.y* blockDim.y));
int kidx=0;
   for(int r=0; r<filterWidth;++r){
        for(int c=0; c<filterWidth;++c){
        
            int idx=(y*s+r)*numCols+x*s+c;
            
        float filter_value=filter[kidx++];
        sum+=filter_value*static_cast<float>(inputChannel[idx]);
   
        }
    }
    outputChannel[thread_1D_pos]=sum;
    
  
  // NOTE: If a thread's absolute position 2D position is within the image, but some of
  // its neighbors are outside the image, then you will need to be extra careful. Instead
  // of trying to read such a neighbor value from GPU memory (which won't work because
  // the value is out of bounds), you should explicitly clamp the neighbor values you read
  // to be within the bounds of the image. If this is not clear to you, then please refer
  // to sequential reference solution for the exact clamping semantics you should follow.
}


//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
  // TODO
  //
  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  // Basically make int2 creates a structure of 2 fields and will initiliaze both of them..

  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);  //  it constructs a vector with value x, y., 
  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;	 //  
  const int absolute_image_position_x = thread_2D_pos.x;
  const int absolute_image_position_y = thread_2D_pos.y;
  if ( absolute_image_position_x >= numCols ||
       absolute_image_position_y >= numRows )
  {
      return;
  }
  redChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].x;
  greenChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].y;
  blueChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].z;
}

//This kernel takes in three color channels and recombines them
//into one image.  The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  //make sure we don't try and access memory outside the image
  //by having any threads mapped there return early
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
//if(thread_1D_pos==500)
	//printf("in kernel value.x=%f\t,value.y=%f\tvalue.z=%f",static_cast<float>(outputImageRGBA[500].x),static_cast<float>(outputImageRGBA[500].y),static_cast<float>(outputImageRGBA[500].z));
}



unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  //allocate memory for the three different channels
  //original
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  //TODO:
  //Allocate memory for the filter on the GPU
  //Use the pointer d_filter that we have already declared for you
  //You need to allocate memory for the filter with hipMalloc
  //be sure to use checkCudaErrors like the above examples to
  //be able to tell if anything goes wrong
  //IMPORTANT: Notice that we pass a pointer to a pointer to hipMalloc
  checkCudaErrors(hipMalloc(&d_filter, sizeof( float) * filterWidth * filterWidth));
  //TODO:
  //Copy the filter on the host (h_filter) to the memory you just allocated
  //on the GPU.  hipMemcpy(dst, src, numBytes, hipMemcpyHostToDevice);
  //Remember to use checkCudaErrors!
  checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));

}


void your_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redBlurred,
                        unsigned char *d_greenBlurred,
                        unsigned char *d_blueBlurred,
                        const int filterWidth,const int tilesize,const int s,const int oRow,const int oCol)
{
  //TODO: Set reasonable block size (i.e., number of threads per block)
  const dim3 blockSize(32,2);


  //TODO:
  //Compute correct grid size (i.e., number of blocks per kernel launch)
  //from the image size and and block size.
  const dim3 gridSize(oCol/blockSize.x+1,oRow/blockSize.y+1);

  /*TODO: Launch a kernel for separating the RGBA image into different color channels
  separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA,
                                              numRows,
                                              numCols,
                                              d_red,
                                              d_green,
                                              d_blue);
  // Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());*/

  //TODO: Call your convolution kernel here 3 times, once for each color channel.
  gaussian_blur<<<gridSize, blockSize>>>(d_red,
                                         d_redBlurred,
                                         numRows,
                                         numCols,
                                         d_filter,
                                         filterWidth,s,oRow,oCol);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  gaussian_blur<<<gridSize, blockSize>>>(d_green,
                                         d_greenBlurred,
                                         numRows,
                                         numCols,
                                         d_filter,
                                         filterWidth,s,oRow,oCol);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  gaussian_blur<<<gridSize, blockSize>>>(d_blue,
                                         d_blueBlurred,
                                         numRows,
                                         numCols,
                                         d_filter,
                                         filterWidth,s,oRow,oCol);

  // Again, call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Now we recombine your results. We take care of launching this kernel for you.
  //
  // NOTE: This kernel launch depends on the gridSize and blockSize variables,
  // which you must set yourself.
  recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                             d_greenBlurred,
                                             d_blueBlurred,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}




void seperate_channel(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                      const size_t numRows, const size_t numCols,
                      const int s,const int oRow,const int oCol)
{
	 
	const dim3 blockSize(32,2);
	const dim3 gridSize(oCol/blockSize.x+1,oRow/blockSize.y+1);


  //TODO: Launch a kernel for separating the RGBA image into different color channels
	separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA,
                                              numRows,
                                              numCols,
                                              d_red,
                                              d_green,
                                              d_blue);

	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
	
}



void conv_firstlayer()
{





}


















